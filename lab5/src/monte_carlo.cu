#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <fstream>
#include <iomanip>
using namespace std;

#define R 1.0
#define PI 3.14159265358979323846


template<typename T> 
__global__ void monte_carlo(unsigned int * map,long long n,T*random_x,T*random_y){
    unsigned int ix= threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = blockIdx.y;
    if (ix < n && iy < n){
        unsigned int ind = iy*n + ix;
        //printf("\n%d ",ind);
        if(sqrt(pow(random_x[ind],2)+pow(random_y[ind],2)<=R)){
            map[ind]=true;
        }
        else
            map[ind]=false;
    }   
}

template<typename T>
__global__ void stretch(T* arr,long long n,T el0,T a){
    unsigned int ix= threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = blockIdx.y;
    if (ix < n && iy < n){
        unsigned int ind = iy*n + ix;
        arr[ind]=arr[ind]*a+el0;
        //printf("%d %lf ",ind ,arr[ind]);
    }   
}

template<typename T>
__global__ void kernel_buffon(unsigned int * map,long long n,T*random_d,T*random_alpha,T l){
    unsigned int ix= threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = blockIdx.y;
    if (ix < n && iy < n){
        unsigned int ind = iy*n + ix;
        if(random_d[ind]/2<l*sin(random_alpha[ind])/2){
            map[ind]=true;
        }
        else{
            map[ind]=false;
        }
    }   
}

long double quarter_monte_carlo(long long n){
    srand(time(0));
    dim3 block(1024);
    dim3 grid((n + block.x - 1) / block.x);

    hiprandGenerator_t gen1;
    hiprandGenerator_t gen2;

    float *random_x, *random_y;
    unsigned int * map_gpu;

    hipMalloc((void **)&random_x, n*sizeof(float)); 
    hipMalloc((void **)&random_y, n*sizeof(float)); 
    hipMalloc((void **)&map_gpu, n*sizeof(unsigned int));
    hiprandCreateGenerator(&gen1, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandCreateGenerator(&gen2, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(gen1, rand()); 
    hiprandSetPseudoRandomGeneratorSeed(gen2, rand()); 

    // create random arrays x and y
    hiprandGenerateUniform(gen1, random_x, n); // x
    hiprandGenerateUniform(gen2, random_y, n); // y

    monte_carlo<<<grid,block>>>(map_gpu,n,random_x,random_y); // do paralell 

    thrust::device_ptr<unsigned int> first(map_gpu);
    thrust::device_ptr<unsigned int> last(map_gpu+n);
    long double res=thrust::reduce(first, last);
    hiprandDestroyGenerator(gen1);
    hiprandDestroyGenerator(gen2);

    hipFree(random_x);
    hipFree(random_y);
    hipFree(map_gpu);
    return res/n*4;
}

long double full_monte_carlo(long long n){
    srand(time(0));
    dim3 block(1024);
    dim3 grid((n + block.x - 1) / block.x);

    hiprandGenerator_t gen1;
    hiprandGenerator_t gen2;

    float *random_x, *random_y;
    unsigned int * map_gpu;

    hipMalloc((void **)&random_x, n*sizeof(float)); 
    hipMalloc((void **)&random_y, n*sizeof(float)); 
    hipMalloc((void **)&map_gpu, n*sizeof(unsigned int));
    hiprandCreateGenerator(&gen1, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandCreateGenerator(&gen2, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(gen1, rand()); 
    hiprandSetPseudoRandomGeneratorSeed(gen2, rand()); 

    // create random arrays x and y
    hiprandGenerateUniform(gen1, random_x, n); // x
    hiprandGenerateUniform(gen2, random_y, n); // y
    stretch<<<grid,block>>>(random_x,n,-1.0f,2.0f);
    stretch<<<grid,block>>>(random_y,n,-1.0f,2.0f);
    monte_carlo<<<grid,block>>>(map_gpu,n,random_x,random_y); // do paralell 
 

    thrust::device_ptr<unsigned int> first(map_gpu);
    thrust::device_ptr<unsigned int> last(map_gpu+n);

    long double res=thrust::reduce(first, last)  ;
    //cout<<res<<endl;
    hiprandDestroyGenerator(gen1);
    hiprandDestroyGenerator(gen2);

    hipFree(random_x);
    hipFree(random_y);
    hipFree(map_gpu);
    return res/n*4;
}

long double buffon(long long n,float l,float L){
    srand(time(0));
    dim3 block(1024);
    dim3 grid((n + block.x - 1) / block.x);
    hiprandGenerator_t gen2,gen3;

    float  *random_d,*random_alpha;
    unsigned int * map_gpu;
    
    hipMalloc((void **)&random_d, n*sizeof(float)); 
    hipMalloc((void **)&random_alpha, n*sizeof(float)); 
    hipMalloc((void **)&map_gpu, n*sizeof(unsigned int));
    hiprandCreateGenerator(&gen2, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandCreateGenerator(&gen3, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(gen2, rand()); 
    hiprandSetPseudoRandomGeneratorSeed(gen3, rand()); 

    // create random arrays x and y
    hiprandGenerateUniform(gen2, random_d, n); // y
    hiprandGenerateUniform(gen3, random_alpha, n); // alpha

    stretch<<<grid,block>>>(random_alpha,n,0.0f,float(PI/2));
    hipDeviceSynchronize();    
    stretch<<<grid,block>>>(random_d,n,0.0f,L);
    hipDeviceSynchronize();    

    kernel_buffon<<<grid,block>>>(map_gpu,n,random_d,random_alpha,l); // do paralell 
    hipDeviceSynchronize();    


    thrust::device_ptr<unsigned int> first(map_gpu);
    thrust::device_ptr<unsigned int> last(map_gpu+n);

    long double res=thrust::reduce(first, last);
    hiprandDestroyGenerator(gen3);
    hiprandDestroyGenerator(gen2);

    hipFree(random_d);
    hipFree(random_alpha);
    hipFree(map_gpu);
    return 2.0f*l*float(n)/res/L;
}

template<typename T> 
void write(T* err,long long * N,int n,string filename){
    ofstream out;
    string name="../TextFiles/"+filename+".txt";
	out.open(name);
	if (out.is_open()) {
        for (int k = 0; k <n; k++){
            out << setprecision(20) << N[k] << " " << setprecision(20) << err[k] << endl;
        }
	}
	else {cout<<"Error while writing"<<endl;}
	out.close();
}

template<typename T> 
void write(T* err,T * N,int n,string filename){
    ofstream out;
    string name="../TextFiles/"+filename+".txt";
	out.open(name);
	if (out.is_open()) {
        for (int k = 0; k <n; k++){
            out << setprecision(20) << N[k] << " " << setprecision(20) << err[k] << endl;
        }
	}
	else {cout<<"Error while writing"<<endl;}
	out.close();
}

int main(){
    long long n=pow(2,28);
    long long n2=1;
    long long N=n*n2;
    long double pi=0;
    
    printf("number of iterations = %lld\n",N);
    
    for (size_t i = 0; i < n2; i++)
        pi+=quarter_monte_carlo(n);    
    printf("Pi quarter = %.16Lf, error = %.16Lf\n",pi/n2,abs(PI-pi/n2));
    

    
    pi=0;
    for (size_t i = 0; i < n2; i++)
        pi+=full_monte_carlo(n);
    printf("Pi full    = %.16Lf, error = %.16Lf\n",pi/n2,abs(PI-pi/n2));
    
    
    int n3=28;
    long long*N_=new long long[n3-8];
    long double *arr=new long double[n3-8];
    for(int k=0;k<n3-8;k++){
        pi=0;
        n=pow(2,k+8);
        for (size_t i = 0; i < n2; i++)
            pi+=buffon(n,1.0f,1.0f);
        arr[k]=abs(PI-pi/n2);
        N_[k]=n;
    }
    write(arr,N_,n3-8,"buffon1");


    n3=20;
    n=pow(2,20);
    long double *frac=new long double[n3];
    for(int k=0;k<n3;k++){
        pi=0;
        float l=0.5+0.025*k;
        for (size_t i = 0; i < n2; i++)
            pi+=buffon(n,l,1.0f);
        arr[k]=abs(PI-pi/n2);
        frac[k]=l;
    }
    write(arr,frac,n3,"buffon2");

    delete[] N_;
    delete[]arr;
    delete[]frac;

    return 0;
}